// stress_gpu.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdbool.h>

// Global flag - volatile ensures the compiler doesn't cache its value.
volatile bool g_running = true;

__global__ void matrixMul(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

extern "C" {

// Runs the stress test loop indefinitely.
// It allocates three matrices on host and device, and in a loop launches the multiplication kernel.
void stressCudaLoop(int N) {
    size_t size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    
    // Allocate host memory.
    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);
    
    // Allocate device memory.
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Initialize input matrices.
    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0f;
        B[i] = 1.0f;
    }
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    printf("Starting GPU stress test loop...\n");

    // Loop until a stop signal is received.
    while (g_running) {
        matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();  // Ensure the kernel has finished before looping again.
    }

    // Optional: Copy back result once after exiting loop.
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free allocated memory.
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("GPU stress test loop ended.\n");
}

// Function to signal the CUDA loop to stop.
void stopStressCuda() {
    g_running = false;
}

} // extern "C"
